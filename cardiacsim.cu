/* 
 * Solves the Panfilov model using an explicit numerical scheme.
 * Based on code orginally provided by Xing Cai, Simula Research Laboratory 
 * and reimplementation by Scott B. Baden, UCSD
 * 
 * Modified and  restructured by Didem Unat, Koc University
 *
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include <iostream>
#include <iomanip>
#include <string.h>
#include <math.h>
#include <sys/time.h>
using namespace std;


static const double kMicro = 1.0e-6;
double getTime()
{
    struct timeval TV;
    struct timezone TZ;

    const int RC = gettimeofday(&TV, &TZ);
    if(RC == -1) {
            printf("ERROR: Bad call to gettimeofday\n");
            return(-1);
    }
    return( ((double)TV.tv_sec) + kMicro * ((double)TV.tv_usec) );
}


double **alloc2D(int m,int n){
   double **E;
   int nx=n, ny=m;
   E = (double**)malloc(sizeof(double*)*ny + sizeof(double)*nx*ny);
   assert(E);
   int j;
   for(j=0;j<ny;j++) 
     E[j] = (double*)(E+ny) + j*nx;
   return(E);
}
    
// Reports statistics about the computation
// These values should not vary (except to within roundoff)
// when we use different numbers of  processes to solve the problem
 double stats(double **E, int m, int n, double *_mx){
     double mx = -1;
     double l2norm = 0;
     int i, j;
     for (j=1; j<=m; j++)
       for (i=1; i<=n; i++) {
	   l2norm += E[j][i]*E[j][i];
	   if (E[j][i] > mx)
	       mx = E[j][i];
      }
     *_mx = mx;
     l2norm /= (double) ((m)*(n));
     l2norm = sqrt(l2norm);
     return l2norm;
 }


void simulate (double** E,  double** E_prev,double** R,
	       const double alpha, const int n, const int m, const double kk,
	       const double dt, const double a, const double epsilon,
	       const double M1,const double  M2, const double b)
{
  int i, j; 
 
  for (j=1; j<=m; j++) 
      E_prev[j][0] = E_prev[j][2];
  for (j=1; j<=m; j++) 
      E_prev[j][n+1] = E_prev[j][n-1];
    
  for (i=1; i<=n; i++) 
      E_prev[0][i] = E_prev[2][i];
  for (i=1; i<=n; i++) 
      E_prev[m+1][i] = E_prev[m-1][i];
    
    // Solve for the excitation, the PDE
    for (j=1; j<=m; j++){
        for (i=1; i<=n; i++) {
	          E[j][i] = E_prev[j][i]+alpha*(E_prev[j][i+1]+E_prev[j][i-1]-4*E_prev[j][i]+E_prev[j+1][i]+E_prev[j-1][i]);
        }
    }
    

  for (j=1; j<=m; j++){
      for (i=1; i<=n; i++)
	        E[j][i] = E[j][i] -dt*(kk* E[j][i]*(E[j][i] - a)*(E[j][i]-1)+ E[j][i] *R[j][i]);
  }
    
    for (j=1; j<=m; j++){
      for (i=1; i<=n; i++)
	        R[j][i] = R[j][i] + dt*(epsilon+M1* R[j][i]/( E[j][i]+M2))*(-R[j][i]-kk* E[j][i]*(E[j][i]-b-1));
    }
    
}

// Main program
int main (int argc, char** argv)
{
  double **E, **R, **E_prev;
  
  // Various constants - these definitions shouldn't change
  const double a=0.1, b=0.1, kk=8.0, M1= 0.07, M2=0.3, epsilon=0.01, d=5e-5;
  
  double T=1000.0;
  int m=200,n=200;

  m = n;  
  E = alloc2D(m+2,n+2);
  E_prev = alloc2D(m+2,n+2);
  R = alloc2D(m+2,n+2);
  
  int i,j;
  // Initialization
  for (j=1; j<=m; j++)
    for (i=1; i<=n; i++)
      E_prev[j][i] = R[j][i] = 0;
  
  for (j=1; j<=m; j++)
    for (i=n/2+1; i<=n; i++)
      E_prev[j][i] = 1.0;
  
  for (j=m/2+1; j<=m; j++)
    for (i=1; i<=n; i++)
      R[j][i] = 1.0;
  
  double dx = 1.0/n;

  // For time integration, these values shouldn't change 
  double rp= kk*(b+1)*(b+1)/4;
  double dte=(dx*dx)/(d*4+((dx*dx))*(rp+kk));
  double dtr=1/(epsilon+((M1/M2)*rp));
  double dt = (dte<dtr) ? 0.95*dte : 0.95*dtr;
  double alpha = d*dt/(dx*dx);

  
  double t0 = getTime();
  double t = 0.0;
  int niter=0;

  while (t<T) {
    
    t += dt;
    niter++;
 
    simulate(E, E_prev, R, alpha, n, m, kk, dt, a, epsilon, M1, M2, b); 
    //swap current E with previous E
    double **tmp = E; E = E_prev; E_prev = tmp;
  }

  double time_elapsed = getTime() - t0;
  cout << "Elapsed Time (sec)          : " << time_elapsed << endl;

  FILE * fp = fopen("v0.txt","w");
  for (int i=1; i<m+1; i++)
      for (int j=1; j<n+1; j++)
          fprintf(fp,"%f\n", E[i][j]);
  fclose(fp);
  
  free (E);
  free (E_prev);
  free (R);
  
  return 0;
}
